/*
    PSPD 2023 - 1
    Alunos: Antonio Aldisio        202028211
            Fernando Miranda Calil 190106565
            Lorrany Oliveira Souza 180113992

    Como compilar:
    $ nvcc jogodavida.cu -o cuda

    Como rodar:
    $ ./cuda

*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#define ind2d(i,j) (i)*(tam+2)+j
#define POWMIN 3
#define POWMAX 10

double wall_time(void) {
  struct timeval tv;
  struct timezone tz;

  gettimeofday(&tv, &tz);
  return(tv.tv_sec + tv.tv_usec/1000000.0);
}

__global__ void UmaVida(int* tabulIn, int* tabulOut, int tam) {
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int vizviv;

  if (i <= tam && j <= tam) {
    vizviv = 	tabulIn[ind2d(i-1,j-1)] + tabulIn[ind2d(i-1,j  )] +
      tabulIn[ind2d(i-1,j+1)] + tabulIn[ind2d(i  ,j-1)] +
      tabulIn[ind2d(i  ,j+1)] + tabulIn[ind2d(i+1,j-1)] +
      tabulIn[ind2d(i+1,j  )] + tabulIn[ind2d(i+1,j+1)];
    if (tabulIn[ind2d(i,j)] && vizviv < 2)
      tabulOut[ind2d(i,j)] = 0;
    else if (tabulIn[ind2d(i,j)] && vizviv > 3)
      tabulOut[ind2d(i,j)] = 0;
    else if (!tabulIn[ind2d(i,j)] && vizviv == 3)
      tabulOut[ind2d(i,j)] = 1;
    else
      tabulOut[ind2d(i,j)] = tabulIn[ind2d(i,j)];
  }
}

void DumpTabul(int * tabul, int tam, int first, int last, char* msg){
  int i, ij;

  printf("%s; Dump posicoes [%d:%d, %d:%d] de tabuleiro %d x %d\n", \
         msg, first, last, first, last, tam, tam);
  for (i=first; i<=last; i++) printf("="); printf("=\n");
  for (i=ind2d(first,0); i<=ind2d(last,0); i+=ind2d(1,0)) {
    for (ij=i+first; ij<=i+last; ij++)
      printf("%c", tabul[ij]? 'X' : '.');
    printf("\n");
  }
  for (i=first; i<=last; i++) printf("="); printf("=\n");
}

void InitTabul(int* tabulIn, int* tabulOut, int tam){
  int ij;

  for (ij=0; ij<(tam+2)*(tam+2); ij++) {
    tabulIn[ij] = 0;
    tabulOut[ij] = 0;
  }

  tabulIn[ind2d(1,2)] = 1; tabulIn[ind2d(2,3)] = 1;
  tabulIn[ind2d(3,1)] = 1; tabulIn[ind2d(3,2)] = 1;
  tabulIn[ind2d(3,3)] = 1;
}

int Correto(int* tabul, int tam){
  int ij, cnt;

  cnt = 0;
  for (ij=0; ij<(tam+2)*(tam+2); ij++)
    cnt = cnt + tabul[ij];
  return (cnt == 5 && tabul[ind2d(tam-2,tam-1)] &&
          tabul[ind2d(tam-1,tam  )] && tabul[ind2d(tam  ,tam-2)] &&
          tabul[ind2d(tam  ,tam-1)] && tabul[ind2d(tam  ,tam  )]);
}

int main(void) {
  int pow;
  int i, tam, *tabulIn, *tabulOut, *d_tabulIn, *d_tabulOut;
  char msg[9];
  double t0, t1, t2, t3;

  for (pow=POWMIN; pow<=POWMAX; pow++) {
    tam = 1 << pow;
    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((tam + threadsPerBlock.x - 1) / threadsPerBlock.x,
                  (tam + threadsPerBlock.y - 1) / threadsPerBlock.y);

    t0 = wall_time();
    tabulIn  = (int *) malloc ((tam+2)*(tam+2)*sizeof(int));
    tabulOut = (int *) malloc ((tam+2)*(tam+2)*sizeof(int));
    InitTabul(tabulIn, tabulOut, tam);
    t1 = wall_time();

    hipMalloc((void**)&d_tabulIn, (tam+2)*(tam+2)*sizeof(int));
    hipMalloc((void**)&d_tabulOut, (tam+2)*(tam+2)*sizeof(int));

    hipMemcpy(d_tabulIn, tabulIn, (tam+2)*(tam+2)*sizeof(int), hipMemcpyHostToDevice);

    for (i=0; i<2*(tam-3); i++) {
      UmaVida<<<numBlocks, threadsPerBlock>>>(d_tabulIn, d_tabulOut, tam);
      hipDeviceSynchronize();
      UmaVida<<<numBlocks, threadsPerBlock>>>(d_tabulOut, d_tabulIn, tam);
      hipDeviceSynchronize();
    }

    hipMemcpy(tabulIn, d_tabulIn, (tam+2)*(tam+2)*sizeof(int), hipMemcpyDeviceToHost);

    if (Correto(tabulIn, tam))
      printf("**RESULTADO CORRETO**\n");
    else
      printf("**RESULTADO ERRADO**\n");

    t2 = wall_time();
    t3 = wall_time();
    printf("tam=%d; tempos: init=%7.7f, comp=%7.7f, fim=%7.7f, tot=%7.7f \n",
           tam, t1-t0, t2-t1, t3-t2, t3-t0);

    hipFree(d_tabulIn);
    hipFree(d_tabulOut);
    free(tabulIn);
    free(tabulOut);
  }
  return 0;
}
